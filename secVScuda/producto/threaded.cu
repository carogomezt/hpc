#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define a 2
#define b 3
#define c 2

void fillMatrix(double *w, int li, int lj){
  double count = 0;
  for(int i=0; i<li; i++){
    for(int j=0; j<lj; j++){
      w[i*lj+j] = count;
      count++;
    }
  }
}

void print(double *w, int li, int lj){
  for(int i=0; i<li; i++){
    for(int j=0; j<lj; j++){
      printf("%.4lf ", w[i*lj+j]);
    }
    printf("\n");
  }
}

__global__
void product(double *d_x, double *d_y, double *d_z){

  int row = blockIdx.y*blockDim.y+threadIdx.y;
  int col = blockIdx.x*blockDim.x+threadIdx.x;
  double sum = 0;
  if ((row < a) && (col < c)){
    for (int i = 0; i < b; i++) sum += d_x[b*row + i] * d_y[i*c+col];
    d_z[row*c+col] = sum;
  }
}

int main(int argc, char const *argv[])
{
  int size1 = a*b*sizeof(double);
  int size2 = b*c*sizeof(double);
  int size3 = a*c*sizeof(double);

  double *x = (double*)malloc(size1);
  double *y = (double*)malloc(size2);
  double *z = (double*)malloc(size3);

  fillMatrix(x,a,b);
  fillMatrix(y,b,c);

  clock_t begin, end;
  double time_spent;
  begin = clock();

  double *d_x = (double*)malloc(size1);
  double *d_y = (double*)malloc(size2);
  double *d_z = (double*)malloc(size3);

  hipMalloc((void**)&d_x, size1);
  hipMalloc((void**)&d_y, size2);
  hipMalloc((void**)&d_z, size3);

  hipMemcpy(d_x, x, size1, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size2, hipMemcpyHostToDevice);

  int threads = 16;
  dim3 dimBlock(threads,threads);
  dim3 dimGrid((c+dimBlock.x-1)/dimBlock.x, (a+dimBlock.y-1)/dimBlock.y);

  product<<<dimGrid,dimBlock>>>(d_x, d_y, d_z);

  hipMemcpy(z,d_z,size3,hipMemcpyDeviceToHost);

  print(x,a,b);
  printf("\n");
  print(y,b,c);
  printf("\n");
  print(z,a,c);

  end = clock();
  time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
  printf("%lf\n", time_spent);

  return 0;
}
